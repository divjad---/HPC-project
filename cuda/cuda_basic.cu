#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "../libs/hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../libs/stb_image.h"
#include "../libs/stb_image_write.h"

// Default values if not set with arguments
int K = 32;
int MAX_ITER = 20;
int BLOCK_SIZE = 256;

void init_clusters_random(unsigned char *imageIn, float *centroids, int width, int height, int cpp) {
    int index;
    int num_pixels = width * height;
    for (int i = 0; i < K; i++) {
        index = rand() % num_pixels;
        for(int j = 0; j < cpp; j++){
            centroids[i * cpp + j] = (float) (imageIn[index * cpp + j]);
        }
    }
}

__global__ void assignPixelsToNearestCentroids(unsigned char *imageIn, int *pixel_cluster_indices, float *centroids, int width, int height, int cpp, int K) {

    int tid = blockIdx.x * blockDim.x+ threadIdx.x;
    int i = tid / width;
    int j = tid % width;

    // Find nearest centroid for each pixel
    if ( i < height && j < width){
        int index = (i * width + j) * cpp;
        int min_cluster_index = 0;
        float min_distance = 1e5;

        for (int cluster = 0; cluster < K; cluster++) {
            float curr_distance = 0;

            for (int channel = 0; channel < cpp; channel++) {
                float diff = ((float)imageIn[index + channel] - centroids[cluster * cpp + channel]);
                curr_distance += diff * diff;
            }

            if (curr_distance < min_distance) {
                min_cluster_index = cluster;
                min_distance = curr_distance;
            }
        }
        pixel_cluster_indices[i * width + j] = min_cluster_index;
    }
}

// SLOWER than atomics in shared memory
__global__ void sumCentroidPositions(unsigned char *imageIn, int *pixel_cluster_indices, int *centroids_sums, int* elements_per_clusters, int width, int height, int cpp) {

    int tid = blockIdx.x * blockDim.x+ threadIdx.x;
    int i = tid / width;
    int j = tid % width;

    // Over each pixel
    if ( i < height && j < width){
        int index = i * width + j;
        int cluster = pixel_cluster_indices[index];

        for (int channel = 0; channel < cpp; channel++) {
            atomicAdd(&centroids_sums[cluster * cpp + channel], (float)imageIn[index * cpp + channel]);
        }

        atomicAdd(&elements_per_clusters[cluster], 1);
    }
}

// SHARED ATOMICS-> this works if K*cpp is less than block size - method below is general, that works in every scenario
__global__ void sumCentroidPositionsSharedMemory(unsigned char *imageIn, int *pixel_cluster_indices, int *centroids_sums, int* elements_per_clusters, int width, int height, int cpp, int K) {

    extern __shared__ int sdata[]; // Shared memory for partial sums
    int *sdata_elements = (int*)&sdata[K * cpp]; // Shared memory for number of elements per cluster
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = tid / width;
    int j = tid % width;

    // Initialize shared memory
    if (threadIdx.x < K * cpp) {
        sdata[threadIdx.x] = 0;
    }

    if (threadIdx.x < K) {
        sdata_elements[threadIdx.x] = 0;
    }
    __syncthreads();

    // Iterate over each pixel
    if (i < height && j < width) {
        int index = i * width + j;
        int cluster = pixel_cluster_indices[index];

        for (int channel = 0; channel < cpp; channel++) {
            atomicAdd(&sdata[cluster * cpp + channel], imageIn[index * cpp + channel]);
        }
        atomicAdd(&sdata_elements[cluster], 1);

    }
    __syncthreads();

    // Update clusters and counts in global memory
    if (threadIdx.x < K * cpp) {
        atomicAdd(&centroids_sums[threadIdx.x], sdata[threadIdx.x]);
    }
    if (threadIdx.x < K) {
        atomicAdd(&elements_per_clusters[threadIdx.x], sdata_elements[threadIdx.x]);
    }
}

//GENERAL SOLUTION - WORKS IN EVERY SCENARIO: K * cpp can be > block size
__global__ void sumCentroidPositionsSharedMemoryWOConstraints(unsigned char *imageIn, int *pixel_cluster_indices, int *centroids_sums, int* elements_per_clusters, int width, int height, int cpp, int K) {

    extern __shared__ int sdata[]; // Shared memory for partial sums
    int *sdata_elements = (int*)&sdata[K * cpp]; // Shared memory for number of elements per cluster
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = tid / width;
    int j = tid % width;

    // Initialize shared memory
    for (int idx = threadIdx.x; idx < K * cpp; idx += blockDim.x) {
        sdata[idx] = 0;
    }

    for (int idx = threadIdx.x; idx < K; idx += blockDim.x) {
        sdata_elements[idx] = 0;
    }
    __syncthreads();

    // Iterate over each pixel
    if (i < height && j < width) {
        int index = i * width + j;
        int cluster = pixel_cluster_indices[index];

        for (int channel = 0; channel < cpp; channel++) {
            atomicAdd(&sdata[cluster * cpp + channel], imageIn[index * cpp + channel]);
        }
        atomicAdd(&sdata_elements[cluster], 1);
    }

    __syncthreads();

    // Update clusters and counts in global memory
    for (int idx = threadIdx.x; idx < K * cpp; idx += blockDim.x) {
        atomicAdd(&centroids_sums[idx], sdata[idx]);
    }

    for (int idx = threadIdx.x; idx < K; idx += blockDim.x) {
        atomicAdd(&elements_per_clusters[idx], sdata_elements[idx]);
    }
}

__device__ int getRandomInteger(int lower, int upper, unsigned int seed) {
    hiprandState state;
    hiprand_init(seed, 0, 0, &state);

    float randomValue = hiprand_uniform(&state);
    return static_cast<int>(randomValue * (upper - lower + 1)) + lower;
}

__global__ void updateCentroidPositions(unsigned char *imageIn, float *centroids, int* centroids_sums, int* elements_per_clusters, int width, int height, int cpp, int K) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Update each centroid position by calculating the average channel value
    if (tid < K * cpp) {
        int cluster = tid / cpp;
        int channel = tid % cpp;

        if (elements_per_clusters[cluster] > 0) {
            centroids[tid] = ((float)centroids_sums[tid]) / elements_per_clusters[cluster];
        } else {
            // Assign random pixel to empty centroid
            unsigned int seed = cluster;
            int random_pixel_i = getRandomInteger(0, width * height - 1, seed);
            centroids[tid] = imageIn[random_pixel_i * cpp + channel];
        }

        centroids_sums[tid] = 0;
        if(channel == 0)
            elements_per_clusters[cluster] = 0;
    }
}

__global__ void mapPixelsToCentroidValues(unsigned char *imageIn, int *pixel_cluster_indices, float *centroids, int width, int height, int cpp, int K) {

    int tid = blockIdx.x * blockDim.x+ threadIdx.x;
    int i = tid / width;
    int j = tid % width;

    // Iterate over each pixel
    if ( i < height && j < width){
        int index = i * width + j;
        int cluster = pixel_cluster_indices[index];

        for (int channel = 0; channel < cpp; channel++) {
            imageIn[index * cpp + channel] = (unsigned char) centroids[cluster * cpp + channel];
        }
    }
}

void kmeans_image_compression(unsigned char *h_image, int width, int height, int cpp, char *image_file) {

    // Create CUDA events and start recording
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipEvent_t iteration_start, iteration_stop;
    hipEventCreate(&iteration_start);
    hipEventCreate(&iteration_stop);

    // Set block and grid sizes
    const size_t blockSize = BLOCK_SIZE;
    const size_t gridSize = (width * height + blockSize - 1) / blockSize;

    // Intialize clusters
    float *h_centroids = (float *) calloc(cpp * K, sizeof(float));
    init_clusters_random(h_image, h_centroids, width, height, cpp);

    // Copy data to GPU
    unsigned char *d_image;
    float *d_centroids;
    int *d_centroids_sums;
    int *d_pixel_cluster_indices;
    int *d_elements_per_cluster;

    checkCudaErrors(hipMalloc(&d_image, width * height * cpp * sizeof(unsigned char)));
    checkCudaErrors(hipMalloc(&d_centroids, K * cpp * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_centroids_sums, K * cpp * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_pixel_cluster_indices, width * height * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_elements_per_cluster, K * sizeof(int)));

    checkCudaErrors(hipMemcpy(d_image, h_image, width * height * cpp * sizeof(unsigned char), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_centroids, h_centroids, K * cpp * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_centroids_sums, 0,  K * cpp * sizeof(int)));
    checkCudaErrors(hipMemset(d_elements_per_cluster, 0, K  * sizeof(int)));
    getLastCudaError("Error while copying data to GPU\n");

    int shared_memory_size = (K * cpp + K) * sizeof(int);

    // Main loop
    printf("Iteration times: [");
    for (int iteration = 0; iteration < MAX_ITER; iteration++) {
        hipEventRecord(iteration_start);
        assignPixelsToNearestCentroids<<<gridSize, blockSize>>>(d_image, d_pixel_cluster_indices, d_centroids, width, height, cpp, K);
        getLastCudaError("Error while assigning pixels to nearest centroids\n");

        sumCentroidPositions<<<gridSize, blockSize, shared_memory_size>>>(d_image, d_pixel_cluster_indices, d_centroids_sums, d_elements_per_cluster, width, height, cpp);
        // sumCentroidPositionsSharedMemoryWOConstraints<<<gridSize, blockSize, shared_memory_size>>>(d_image, d_pixel_cluster_indices, d_centroids_sums, d_elements_per_cluster, width, height, cpp, K);
        getLastCudaError("Error while summation of centroid vales\n");

        updateCentroidPositions<<<((K * cpp + BLOCK_SIZE -1)/BLOCK_SIZE), BLOCK_SIZE>>>(d_image, d_centroids, d_centroids_sums, d_elements_per_cluster, width, height, cpp, K);
        getLastCudaError("Error while updating positions of centroids\n");
        hipEventRecord(iteration_stop);
        hipEventSynchronize(iteration_stop);
        if (iteration > 0) {
            printf(", ");
        }
        float milis = 0.0f;
        hipEventElapsedTime(&milis, iteration_start, iteration_stop);
        printf("%f", milis);
    }
    printf("]\n");
    // Assign pixels to final clusters
    mapPixelsToCentroidValues<<<gridSize, blockSize>>>(d_image, d_pixel_cluster_indices, d_centroids, width, height, cpp, K);

    // Save the compreesed image
    checkCudaErrors(hipMemcpy(h_image, d_image, width * height * cpp * sizeof(unsigned char), hipMemcpyDeviceToHost));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time: %.4f \n", milliseconds);

    char output_file[256];
    strcpy(output_file, image_file);
    char *extension = strrchr(output_file, '.');
    if (extension != NULL) *extension = '\0';  // Cut off the file extension
    strcat(output_file, "_compressedGPU.png");
    stbi_write_png(output_file, width, height, cpp, h_image, width * cpp);

    hipFree(d_image);
    hipFree(d_centroids);
    hipFree(d_centroids_sums);
    hipFree(d_pixel_cluster_indices);
    hipFree(d_elements_per_cluster);
}

int main(int argc, char **argv)
{
    if (argc < 2){
        fprintf(stderr, "Not enough arguments\n");
        exit(1);
    }
    srand(42);
    char *image_file = argv[1];
    if (argc > 2) BLOCK_SIZE = atoi(argv[2]);
    if (argc > 3) K = atoi(argv[3]);
    if (argc > 4) MAX_ITER = atoi(argv[4]);

    int width, height, cpp;
    unsigned char *h_image = stbi_load(image_file, &width, &height, &cpp, 0);

    if(!h_image) return 0;

    kmeans_image_compression(h_image, width, height, cpp, image_file);

    stbi_image_free(h_image);
}